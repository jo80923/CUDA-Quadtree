#include "hip/hip_runtime.h"
#include "ImageTree.cuh"



/*
CUDA KERNELS
*/

__global__ void generateLeafNodes(Node* leafNodes, unsigned int width, uint2 imageSize, int depth){
  int globalID = blockIdx.x *blockDim.x + threadIdx.x;
  if(globalID < imageSize.x*imageSize.y){
    int x = globalID%imageSize.x;
    int y = globalID/imageSize.x;
    if(imageSize.x > imageSize.y){
      x += width/2;
    }
    else{
      y += width/2;
    }
    int key = 0;
    unsigned int depth_reg = depth;
    int currentDepth = 1;
    int W = width/2;
    int2 center = {W,W};
    while(W > 1 && depth_reg != currentDepth){
      W /= 2;
      if(x < center.x){
        key <<= 1;
        center.x -= W;
      }
      else{
        key = (key << 1) + 1;
        center.x += W;
      }
      if(y < center.y){
        key <<= 1;
        center.y -= W;
      }
      else{
        key = (key << 1) + 1;
        center.y += W;
      }
      currentDepth++;
    }
    Node leaf = Node();
    leaf.key = key;
    leaf.center = {center.x + 0.5f, center.y + 0.5f};
    leaf.depth = currentDepth;
    leafNodes[globalID] = leaf;
  }
}

/*
CLASS AND STRUCT METHODS
*/
template<typename T>
__device__ __host__ ImageTree<T>::Node::Node(){
  this->key = -1;
  this->dataIndex = -1;
  this->numElements = 0;
  this->center = {-1,-1};
  this->depth = -1;
  this->parent = -1;
  for(int i = 0; i < 4; ++i) this->children[i] = -1;
  for(int i = 0; i < 9; ++i) this->neighbors[i] = -1;
  for(int i = 0; i < 4; ++i) this->edges[i] = -1;
  for(int i = 0; i < 4; ++i) this->vertices[i] = -1;
}
template<typename T>
__device__ __host__ ImageTree<T>::Vertex::Vertex(){
  this->loc = {-1,-1};
  for(int i = 0; i < 4; ++i) this->nodes[i] = -1;
  this->depth = -1;
}
template<typename T>
__device__ __host__ ImageTree<T>::Edge::Edge(){
  this->vertices = {-1,-1};
  for(int i = 0; i < 2; ++i) this->nodes[i] = -1;
  this->depth = -1;
}

template<typename T>
ImageTree<T>::ImageTree(){
  this->imageSize = {0,0};
  this->nodes = nullptr;
  this->data = nullptr;
  this->edges = nullptr;
  this->vertices = nullptr;
  this->imageSize = {0,0};
}
template<typename T>
ImageTree<T>::ImageTree(uint2 imageSize, jax::Unity<T>* data){
  this->nodes = nullptr;
  this->edges = nullptr;
  this->vertices = nullptr;
  this->data = data;
  this->imageSize = imageSize;
  this->width = (imageSize.x > imageSize.y) ? imageSize.x : imageSize.y;
  if(width % 2 != 0) this->width += 1;
  this->generateLeafNodes();
  this->generateParentNodes();
}


template<typename T>
ImageTree<T>::~ImageTree(){
  if(this->nodes != nullptr) delete this->nodes;
  if(this->vertices != nullptr) delete this->vertices;
  if(this->edges != nullptr) delete this->edges;
  if(this->data != nullptr) delete this->data;
  if(this->nodeDepthIndex != nullptr) delete this->nodeDepthIndex;
  if(this->vertexDepthIndex != nullptr) delete this->vertexDepthIndex;
  if(this->edgeDepthIndex != nullptr) delete this->edgeDepthIndex;
}


template<typename T>
void ImageTree<T>::generateLeafNodes(int depth){
  Node* leafNodes_device = nullptr;
  CudaSafeCall(hipMalloc((void**)&leafNodes_device, this->data->numElements*sizeof(Node)));
  dim3 grid = {(this->data->numElements/1024) + 1,1,1};
  dim3 block = {1024,1,1};
  generateLeafNodes<<<grid,block>>>(leafNodes_device, this->width, this->imageSize, depth);
  CudaCheckError();
  this->nodes = new Unity<Node>(leafNodes_device, this->data->numElements, jax::gpu);
  if(depth != -1){
    //TO USE IN POINT BASED QUADTREE

    // thrust::counting_iterator<unsigned int> iter(0);
    // thrust::device_vector<unsigned int> indices(this->data->numElements);
    // thrust::copy(iter, iter + this->data->numElements, indices.begin());
    //
    // unsigned int* nodePointIndex = new unsigned int[this->points->numElements]();
    // CudaSafeCall(hipMemcpy(nodePointIndex, thrust::raw_pointer_cast(indices.data()), this->data->numElements*sizeof(unsigned int),hipMemcpyDeviceToHost));
    //
    // thrust::device_ptr<int> kys(nodeKeys_device);
    // thrust::sort_by_key(kys, kys + this->data->numElements, indices.begin());
    //
    // if(this->data->fore != jax::gpu){
    //   this->data->transferMemoryTo(jax::gpu);
    // }
    //
    // thrust::device_ptr<float2> cnts(nodeCenters_device);
    // thrust::device_vector<float2> sortedCnts(this->data->numElements);
    // thrust::gather(indices.begin(), indices.end(), cnts, sortedCnts.begin());
    // CudaSafeCall(hipMemcpy(nodeCenters_device, thrust::raw_pointer_cast(sortedCnts.data()), this->data->numElements*sizeof(float2),hipMemcpyDeviceToDevice));
    //
    // thrust::device_ptr<T> dataSorter(this->data->device);
    // thrust::device_vector<T> sortedData(this->data->numElements);
    // thrust::gather(indices.begin(), indices.end(), dataSorter, sortedData.begin());
    // //determine if this is necessary
    // this->data->setData(thrust::raw_pointer_cast(sortedData.data()), this->data->numElements, jax::gpu);
    // this->data->transferMemoryTo(jax::cpu);
    // this->data->clearDevice();
    //
    // //there may be a faster way to do this
    // thrust::pair<int*, unsigned int*> new_end;//the last value of these node array
    // new_end = thrust::unique_by_key(kys,kys + this->data->numElements, indices.begin());

    //now you need to copy over all the nonredudant nodes

  }
}




template<typename T>
void ImageTree<T>::generateParentNodes(){
  if(this->nodes == nullptr || this->nodes->state == jax::null){
    //TODO potentially develop support for bottom up growth
    throw jax::NullUnityException("Cannot generate parent nodes before children");
  }

}
