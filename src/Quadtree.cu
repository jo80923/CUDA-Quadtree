#include "hip/hip_runtime.h"
#include "Quadtree.cuh"

/*
Accepted types for data
-listed to avoid link issues
- IF you want to add an acceptable type to this list do the following
  1. add definition like below
  2. implement getKeys for that data type

*/
//for points
template class jax::Quadtree<float2>;
//for pointers to external data - must be flattened row wise and will be dense (dataIndex = y*dimX + x)
//also requires size of container x,y
template class jax::Quadtree<unsigned int>;
//just image tree
template class jax::Quadtree<unsigned char>;
//for Localized Pointers to external data
template class jax::Quadtree<jax::LocalizedData<unsigned int>>;

/*
CLASS AND STRUCT METHODS
*/

template<typename T>
__device__ __host__ jax::Quadtree<T>::Node::Node(){
  this->key = -1;
  this->dataIndex = -1;
  this->numElements = 0;
  this->center = {-1,-1};
  this->depth = -1;
  this->parent = -1;
  this->flag = false;
  for(int i = 0; i < 4; ++i) this->children[i] = -1;
  for(int i = 0; i < 9; ++i) this->neighbors[i] = -1;
  for(int i = 0; i < 4; ++i) this->edges[i] = -1;
  for(int i = 0; i < 4; ++i) this->vertices[i] = -1;
}
template<typename T>
__device__ __host__ jax::Quadtree<T>::Vertex::Vertex(){
  this->loc = {-1,-1};
  for(int i = 0; i < 4; ++i) this->nodes[i] = -1;
  this->depth = -1;
}
template<typename T>
__device__ __host__ jax::Quadtree<T>::Edge::Edge(){
  this->vertices = {-1,-1};
  for(int i = 0; i < 2; ++i) this->nodes[i] = -1;
  this->depth = -1;
}

template<typename T>
jax::Quadtree<T>::Quadtree(){
  this->nodes = nullptr;
  this->data = nullptr;
  this->edges = nullptr;
  this->vertices = nullptr;
  this->size = {0,0};
  this->border = {0,0};
}

//TODO throw error if depth.x is greater than depth.y
//specifically for index full quadtree
template<>
jax::Quadtree<unsigned int>::Quadtree(uint2 size, unsigned int depth, int2 border){
  this->nodes = nullptr;
  this->edges = nullptr;
  this->vertices = nullptr;
  unsigned int* data_host = new unsigned int[size.x*size.y];
  for(int i = 0; i < size.x*size.y; ++i){
    data_host[i] = i;
  }
  this->data = new Unity<unsigned int>(data_host, size.x*size.y, cpu);
  this->border = border;
  this->size = {size.x + (border.x*2),size.y + (border.y*2)};
  this->depth = depth;
  printf("Building Quadtree with depth = %d\n",this->depth);
  this->generateLeafNodes();
  this->generateParentNodes();
  this->fillNeighborhoods();
}
template<typename T>
jax::Quadtree<T>::Quadtree(uint2 size, unsigned int depth, jax::Unity<T>* data, unsigned int colorDepth, int2 border){
  this->nodes = nullptr;
  this->edges = nullptr;
  this->vertices = nullptr;
  this->data = data;
  this->colorDepth = colorDepth;
  this->border = border;
  this->size = {size.x + (border.x*2),size.y + (border.y*2)};
  this->depth = depth;
  printf("Building Quadtree with following characteristics:\ndepth = %d",this->depth);
  printf("\nsize = {%d,%d}\nborder = {%d,%d}\n",this->size.x,this->size.y,this->border.x,this->border.y);
  this->generateLeafNodes();
  this->generateParentNodes();
  this->fillNeighborhoods();
}

template<typename T>
jax::Quadtree<T>::~Quadtree(){
  if(this->nodes != nullptr) delete this->nodes;
  if(this->vertices != nullptr) delete this->vertices;
  if(this->edges != nullptr) delete this->edges;
  if(this->data != nullptr) delete this->data;
  if(this->nodeDepthIndex != nullptr) delete this->nodeDepthIndex;
  if(this->vertexDepthIndex != nullptr) delete this->vertexDepthIndex;
  if(this->edgeDepthIndex != nullptr) delete this->edgeDepthIndex;
  if(this->dataNodeIndex != nullptr) delete this->dataNodeIndex;
}


//TODO ensure numLeafNodes cant go over max int (conditional usage of gridDim.y)
//check if log2 will work
template<typename T>
void jax::Quadtree<T>::generateLeafNodes(){

  clock_t timer = clock();
  std::cout<<"generating leaf nodes for quadtree..."<<std::endl;
  int* leafNodeKeys_device = nullptr;
  float2* leafNodeCenters_device = nullptr;
  unsigned int* nodeDataIndex_device = nullptr;

  unsigned long numLeafNodes = 0;
  numLeafNodes = this->data->numElements;
  CudaSafeCall(hipMalloc((void**)&leafNodeKeys_device, numLeafNodes*sizeof(int)));
  CudaSafeCall(hipMalloc((void**)&leafNodeCenters_device, numLeafNodes*sizeof(float2)));
  dim3 grid = {1,1,1};
  dim3 block = {1,1,1};
  if(this->colorDepth > 1){
    getGrid(numLeafNodes/this->colorDepth,grid);
    block = {this->colorDepth,1,1};
    getKeys<<<grid,block>>>(leafNodeKeys_device, leafNodeCenters_device, this->size, this->border, this->depth, this->colorDepth);
    hipDeviceSynchronize();
    CudaCheckError();
  }
  else{
    grid = {(numLeafNodes/1024) + 1,1,1};
    block = {1024,1,1};
    getKeys<<<grid,block>>>(leafNodeKeys_device, leafNodeCenters_device, this->size, this->border, this->depth);
    hipDeviceSynchronize();
    CudaCheckError();
  }


  thrust::counting_iterator<unsigned int> iter(0);
  thrust::device_vector<unsigned int> indices(this->data->numElements);
  thrust::copy(iter, iter + this->data->numElements, indices.begin());
  CudaSafeCall(hipMalloc((void**)&nodeDataIndex_device, numLeafNodes*sizeof(unsigned int)));
  CudaSafeCall(hipMemcpy(nodeDataIndex_device, thrust::raw_pointer_cast(indices.data()), numLeafNodes*sizeof(unsigned int),hipMemcpyDeviceToDevice));

  thrust::device_ptr<int> kys(leafNodeKeys_device);
  thrust::sort_by_key(kys, kys + this->data->numElements, indices.begin());

  thrust::device_ptr<float2> cnts(leafNodeCenters_device);
  thrust::device_vector<float2> sortedCnts(this->data->numElements);
  thrust::gather(indices.begin(), indices.end(), cnts, sortedCnts.begin());
  CudaSafeCall(hipMemcpy(leafNodeCenters_device, thrust::raw_pointer_cast(sortedCnts.data()), this->data->numElements*sizeof(float2),hipMemcpyDeviceToDevice));

  if(this->data->fore != gpu){
    this->data->transferMemoryTo(gpu);
  }

  thrust::device_ptr<T> dataSorter(this->data->device);
  thrust::device_vector<T> sortedData(this->data->numElements);
  thrust::gather(indices.begin(), indices.end(), dataSorter, sortedData.begin());
  T* data_device = nullptr;
  CudaSafeCall(hipMalloc((void**)&data_device,this->data->numElements*sizeof(T)));
  CudaSafeCall(hipMemcpy(data_device,thrust::raw_pointer_cast(sortedData.data()), this->data->numElements*sizeof(T), hipMemcpyDeviceToDevice));
  this->data->setData(data_device, this->data->numElements, gpu);
  this->data->transferMemoryTo(cpu);
  this->data->clear(gpu);

  thrust::pair<thrust::device_ptr<int>, thrust::device_ptr<unsigned int>> new_end;//the last value of these node array
  thrust::device_ptr<unsigned int> compactNodeDataIndex(nodeDataIndex_device);
  new_end = thrust::unique_by_key(kys,kys + this->data->numElements, compactNodeDataIndex);
  numLeafNodes = thrust::get<0>(new_end) - kys;

  Node* leafNodes_device = nullptr;
  CudaSafeCall(hipMalloc((void**)&leafNodes_device, numLeafNodes*sizeof(Node)));

  grid = {(numLeafNodes/1024)+1,1,1};
  block = {1024,1,1};

  fillLeafNodes<T><<<grid,block>>>(this->data->numElements, numLeafNodes,leafNodes_device,leafNodeKeys_device,leafNodeCenters_device,nodeDataIndex_device,this->depth);
  hipDeviceSynchronize();
  CudaCheckError();

  this->nodes = new Unity<Node>(leafNodes_device, numLeafNodes, gpu);

  this->nodes->fore = gpu;

  CudaSafeCall(hipFree(leafNodeKeys_device));
  CudaSafeCall(hipFree(leafNodeCenters_device));
  CudaSafeCall(hipFree(nodeDataIndex_device));

  printf("done in %f seconds.\n\n",((float) clock() -  timer)/CLOCKS_PER_SEC);

}


template<typename T>
void jax::Quadtree<T>::generateParentNodes(){
  clock_t timer = clock();
  std::cout<<"filling coarser depths of quadtree..."<<std::endl;
  if(this->nodes == nullptr || this->nodes->state == null){
    //TODO potentially develop support for bottom up growth
    throw NullUnityException("Cannot generate parent nodes before children");
  }
  Node* uniqueNodes_device;
  if(this->nodes->state == cpu){
    this->nodes->transferMemoryTo(gpu);
  }
  int numUniqueNodes = this->nodes->numElements;
  CudaSafeCall(hipMalloc((void**)&uniqueNodes_device, this->nodes->numElements*sizeof(Node)));
  CudaSafeCall(hipMemcpy(uniqueNodes_device, this->nodes->device, this->nodes->numElements*sizeof(Node), hipMemcpyDeviceToDevice));
  delete this->nodes;
  this->nodes = nullptr;
  unsigned int totalNodes = 0;

  Node** nodes2D = new Node*[this->depth + 1];

  unsigned int* nodeAddresses_device;
  unsigned int* nodeNumbers_device;

  unsigned int* nodeDepthIndex_host = new unsigned int[this->depth + 1]();
  this->nodeDepthIndex = new Unity<unsigned int>(nodeDepthIndex_host, this->depth + 1, cpu);

  dim3 grid = {1,1,1};
  dim3 block = {1,1,1};
  getFlatGridBlock(numUniqueNodes, grid, block);

  for(int d = this->depth; d >= 0; --d){

    CudaSafeCall(hipMalloc((void**)&nodeNumbers_device, numUniqueNodes * sizeof(unsigned int)));
    CudaSafeCall(hipMalloc((void**)&nodeAddresses_device, numUniqueNodes * sizeof(unsigned int)));
    //this is just to fill the arrays with 0s

    findAllNodes<T><<<grid,block>>>(numUniqueNodes, nodeNumbers_device, uniqueNodes_device);
    hipDeviceSynchronize();
    CudaCheckError();
    thrust::device_ptr<unsigned int> nN(nodeNumbers_device);
    thrust::device_ptr<unsigned int> nA(nodeAddresses_device);
    thrust::inclusive_scan(nN, nN + numUniqueNodes, nA);

    unsigned int numNodesAtDepth = 0;
    CudaSafeCall(hipMemcpy(&numNodesAtDepth, nodeAddresses_device + (numUniqueNodes - 1), sizeof(unsigned int), hipMemcpyDeviceToHost));
    numNodesAtDepth = (d > 0) ? numNodesAtDepth + 4: 1;

    CudaSafeCall(hipMalloc((void**)&nodes2D[this->depth - d], numNodesAtDepth*sizeof(Node)));
    Node* blankNodes = new Node[numNodesAtDepth]();
    CudaSafeCall(hipMemcpy(nodes2D[this->depth - d], blankNodes, numNodesAtDepth*sizeof(Node),hipMemcpyHostToDevice));
    delete[] blankNodes;

    fillNodesAtDepth<T><<<grid,block>>>(numUniqueNodes, nodeNumbers_device, nodeAddresses_device, uniqueNodes_device, nodes2D[this->depth - d], d, this->depth);
    hipDeviceSynchronize();
    CudaCheckError();
    CudaSafeCall(hipFree(uniqueNodes_device));
    CudaSafeCall(hipFree(nodeAddresses_device));
    CudaSafeCall(hipFree(nodeNumbers_device));

    numUniqueNodes = numNodesAtDepth / 4;
    if(d != 0){
      grid = {1,1,1};
      block = {1,1,1};
      CudaSafeCall(hipMalloc((void**)&uniqueNodes_device, numUniqueNodes*sizeof(Node)));
      getFlatGridBlock(numUniqueNodes, grid, block);
      buildParentalNodes<T><<<grid,block>>>(numNodesAtDepth,totalNodes,nodes2D[this->depth - d],uniqueNodes_device,this->size);
      hipDeviceSynchronize();
      CudaCheckError();
    }
    this->nodeDepthIndex->host[this->depth - d] = totalNodes;
    totalNodes += numNodesAtDepth;
  }
  unsigned int numRootNodes = totalNodes - this->nodeDepthIndex->host[this->depth];
  Node* nodes_device = nullptr;
  CudaSafeCall(hipMalloc((void**)&nodes_device,totalNodes*sizeof(Node)));
  this->nodes = new Unity<Node>(nodes_device, totalNodes, gpu);
  for(int i = 0; i <= this->depth; ++i){
    if(i < this->depth){
      CudaSafeCall(hipMemcpy(this->nodes->device + this->nodeDepthIndex->host[i], nodes2D[i],
        (this->nodeDepthIndex->host[i+1]-this->nodeDepthIndex->host[i])*sizeof(Node), hipMemcpyDeviceToDevice));
    }
    else{
      CudaSafeCall(hipMemcpy(this->nodes->device + this->nodeDepthIndex->host[i],
        nodes2D[i], numRootNodes*sizeof(Node), hipMemcpyDeviceToDevice));
    }
    CudaSafeCall(hipFree(nodes2D[i]));
  }
  delete[] nodes2D;

  unsigned int* dataNodeIndex_device = nullptr;
  CudaSafeCall(hipMalloc((void**)&dataNodeIndex_device, this->data->numElements*sizeof(unsigned int)));
  this->dataNodeIndex = new Unity<unsigned int>(dataNodeIndex_device, this->data->numElements, gpu);

  unsigned int numNodesAtDepth = 1;
  unsigned int depthStartingIndex = 0;
  grid = {1,1,1};
  block = {4,1,1};
  for(int i = this->depth; i >= 0; --i){
    depthStartingIndex = this->nodeDepthIndex->host[i];
    if(i != (int)this->depth){
      numNodesAtDepth = this->nodeDepthIndex->host[i + 1] - depthStartingIndex;
    }
    getGrid(numNodesAtDepth, grid);
    fillParentIndex<T><<<grid, block>>>(numNodesAtDepth, depthStartingIndex, this->nodes->device);
    CudaCheckError();
  }

  grid = {1,1,1};
  block = {1,1,1};
  getFlatGridBlock(this->nodeDepthIndex->host[1],grid,block);
  fillDataNodeIndex<T><<<grid,block>>>(this->nodeDepthIndex->host[1],this->nodes->device, this->dataNodeIndex->device);
  hipDeviceSynchronize();
  CudaCheckError();
  printf("TOTAL NODES = %d\n\n",totalNodes);
  printf("done in %f seconds.\n\n",((float) clock() -  timer)/CLOCKS_PER_SEC);

  this->nodes->fore = gpu;
  this->dataNodeIndex->fore = gpu;
}

template<typename T>
void jax::Quadtree<T>::fillNeighborhoods(){
  unsigned int parentLUT[4][9] = {
    {0,1,1,3,4,4,3,4,4},
    {1,1,2,4,4,5,4,4,5},
    {3,4,4,3,4,4,6,7,7},
    {4,4,5,4,4,5,7,7,8}
  };
  unsigned int childLUT[4][9] = {
    {3,2,3,1,0,1,3,2,3},
    {2,3,2,0,1,0,2,3,2},
    {1,0,1,3,2,3,1,0,1},
    {0,1,0,2,3,2,0,1,0}
  };
  unsigned int* parentLUT_device = nullptr;
  unsigned int* childLUT_device = nullptr;
  CudaSafeCall(hipMalloc((void**)&parentLUT_device, 36*sizeof(int)));
  CudaSafeCall(hipMalloc((void**)&childLUT_device, 36*sizeof(int)));
  for(int i = 0; i < 4; ++i){
    CudaSafeCall(hipMemcpy(parentLUT_device + i*9, &(parentLUT[i]), 9*sizeof(int), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(childLUT_device + i*9, &(childLUT[i]), 9*sizeof(int), hipMemcpyHostToDevice));
  }

  dim3 grid = {1,1,1};
  dim3 block = {9,1,1};

  unsigned int numNodesAtDepth = 0;
  unsigned int depthStartingIndex = 0;
  for(int i = this->depth; i >= 0; --i){
    numNodesAtDepth = 1;
    depthStartingIndex = this->nodeDepthIndex->host[i];
    if(i != this->depth){
      numNodesAtDepth = this->nodeDepthIndex->host[i + 1] - depthStartingIndex;
    }
    getGrid(numNodesAtDepth, grid);
    computeNeighboringNodes<T><<<grid, block>>>(numNodesAtDepth, depthStartingIndex, parentLUT_device, childLUT_device, this->nodes->device);
    hipDeviceSynchronize();
    CudaCheckError();
  }
  this->nodes->fore = gpu;//just to ensure that it is known gpu nodes was edited last
  CudaSafeCall(hipFree(parentLUT_device));
  CudaSafeCall(hipFree(childLUT_device));
  std::cout<<"Neighborhoods filled"<<std::endl;
}

template<typename T>
void jax::Quadtree<T>::generateVertices(){

  unsigned int numNodesAtDepth = 0;
  dim3 grid = {1,1,1};
  dim3 block = {1,1,1};
  int* atomicCounter;
  int numVertices = 0;
  CudaSafeCall(hipMalloc((void**)&atomicCounter, sizeof(int)));
  CudaSafeCall(hipMemcpy(atomicCounter, &numVertices, sizeof(int), hipMemcpyHostToDevice));
  Vertex** vertices2D_device;
  CudaSafeCall(hipMalloc((void**)&vertices2D_device, (this->depth + 1)*sizeof(Vertex*)));
  Vertex** vertices2D = new Vertex*[this->depth + 1];

  unsigned int* vertexDepthIndex_host = new unsigned int[this->depth + 1];

  int prevCount = 0;
  int* ownerInidices_device;
  int* vertexPlacement_device;
  int* compactedOwnerArray_device;
  int* compactedVertexPlacement_device;
  for(int i = 0; i <= this->depth; ++i){
    //reset previously allocated resources
    grid.y = 1;
    block.x = 4;
    if(i == this->depth){//WARNING MAY CAUSE ISSUE
      numNodesAtDepth = this->nodes->numElements - this->nodeDepthIndex->host[this->depth];
    }
    else{
      numNodesAtDepth = this->nodeDepthIndex->host[i + 1] - this->nodeDepthIndex->host[i];
    }

    getGrid(numNodesAtDepth,grid);

    int* ownerInidices = new int[numNodesAtDepth*4];
    for(int v = 0;v < numNodesAtDepth*4; ++v){
      ownerInidices[v] = -1;
    }
    CudaSafeCall(hipMalloc((void**)&ownerInidices_device,numNodesAtDepth*4*sizeof(int)));
    CudaSafeCall(hipMalloc((void**)&vertexPlacement_device,numNodesAtDepth*4*sizeof(int)));
    CudaSafeCall(hipMemcpy(ownerInidices_device, ownerInidices, numNodesAtDepth*4*sizeof(int), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(vertexPlacement_device, ownerInidices, numNodesAtDepth*4*sizeof(int), hipMemcpyHostToDevice));
    delete[] ownerInidices;

    prevCount = numVertices;
    vertexDepthIndex_host[i] = numVertices;

    findVertexOwners<T><<<grid, block>>>(numNodesAtDepth, this->nodeDepthIndex->host[i], this->nodes->device, atomicCounter, ownerInidices_device, vertexPlacement_device);
    CudaCheckError();
    CudaSafeCall(hipMemcpy(&numVertices, atomicCounter, sizeof(int), hipMemcpyDeviceToHost));
    if(i == this->depth  && numVertices - prevCount != 4){
      std::cout<<"ERROR GENERATING VERTICES, vertices at depth 0 != 4 -> "<<numVertices - prevCount<<std::endl;
      exit(-1);
    }

    CudaSafeCall(hipMalloc((void**)&vertices2D[i], (numVertices - prevCount)*sizeof(Vertex)));
    CudaSafeCall(hipMalloc((void**)&compactedOwnerArray_device,(numVertices - prevCount)*sizeof(int)));
    CudaSafeCall(hipMalloc((void**)&compactedVertexPlacement_device,(numVertices - prevCount)*sizeof(int)));

    thrust::device_ptr<int> arrayToCompact(ownerInidices_device);
    thrust::device_ptr<int> arrayOut(compactedOwnerArray_device);
    thrust::device_ptr<int> placementToCompact(vertexPlacement_device);
    thrust::device_ptr<int> placementOut(compactedVertexPlacement_device);

    //TODO change to just remove
    thrust::copy_if(arrayToCompact, arrayToCompact + (numNodesAtDepth*4), arrayOut, is_not_neg());
    CudaCheckError();
    thrust::copy_if(placementToCompact, placementToCompact + (numNodesAtDepth*4), placementOut, is_not_neg());
    CudaCheckError();

    CudaSafeCall(hipFree(ownerInidices_device));
    CudaSafeCall(hipFree(vertexPlacement_device));

    grid = {1,1,1};
    block = {1,1,1};
    getGrid(numVertices - prevCount, grid);

    fillUniqueVertexArray<T><<<grid, block>>>(this->nodeDepthIndex->host[i], this->nodes->device, numVertices - prevCount,
      vertexDepthIndex_host[i], vertices2D[i], this->depth - i, compactedOwnerArray_device, compactedVertexPlacement_device,this->size);
    CudaCheckError();
    CudaSafeCall(hipFree(compactedOwnerArray_device));
    CudaSafeCall(hipFree(compactedVertexPlacement_device));

  }
  Vertex* vertices_device;
  CudaSafeCall(hipMalloc((void**)&vertices_device, numVertices*sizeof(Vertex)));
  for(int i = 0; i <= this->depth; ++i){
    if(i < this->depth){
      CudaSafeCall(hipMemcpy(vertices_device + vertexDepthIndex_host[i], vertices2D[i], (vertexDepthIndex_host[i+1] - vertexDepthIndex_host[i])*sizeof(Vertex), hipMemcpyDeviceToDevice));
    }
    else{
      CudaSafeCall(hipMemcpy(vertices_device + vertexDepthIndex_host[i], vertices2D[i], 4*sizeof(Vertex), hipMemcpyDeviceToDevice));
    }
    CudaSafeCall(hipFree(vertices2D[i]));
  }
  CudaSafeCall(hipFree(vertices2D_device));

  this->vertices = new Unity<Vertex>(vertices_device, numVertices, gpu);
  this->vertexDepthIndex = new Unity<unsigned int>(vertexDepthIndex_host, this->depth + 1, cpu);

}

template<typename T>
void jax::Quadtree<T>::generateEdges(){
  unsigned int numNodesAtDepth = 0;
  dim3 grid = {1,1,1};
  dim3 block = {1,1,1};
  int* atomicCounter;
  int numEdges = 0;
  CudaSafeCall(hipMalloc((void**)&atomicCounter, sizeof(int)));
  CudaSafeCall(hipMemcpy(atomicCounter, &numEdges, sizeof(int), hipMemcpyHostToDevice));
  Edge** edges2D_device;
  CudaSafeCall(hipMalloc((void**)&edges2D_device, (this->depth + 1)*sizeof(Edge*)));
  Edge** edges2D = new Edge*[this->depth + 1];

  unsigned int* edgeDepthIndex_host = new unsigned int[this->depth + 1];

  int prevCount = 0;
  int* ownerInidices_device;
  int* edgePlacement_device;
  int* compactedOwnerArray_device;
  int* compactedEdgePlacement_device;
  for(int i = 0; i <= this->depth; ++i){
    //reset previously allocated resources
    grid.y = 1;
    block.x = 4;
    if(i == this->depth){//WARNING MAY CAUSE ISSUE
      numNodesAtDepth = 1;
    }
    else{
      numNodesAtDepth = this->nodeDepthIndex->host[i + 1] - this->nodeDepthIndex->host[i];
    }

    getGrid(numNodesAtDepth,grid);

    int* ownerInidices = new int[numNodesAtDepth*4];
    for(int v = 0;v < numNodesAtDepth*4; ++v){
      ownerInidices[v] = -1;
    }
    CudaSafeCall(hipMalloc((void**)&ownerInidices_device,numNodesAtDepth*4*sizeof(int)));
    CudaSafeCall(hipMalloc((void**)&edgePlacement_device,numNodesAtDepth*4*sizeof(int)));
    CudaSafeCall(hipMemcpy(ownerInidices_device, ownerInidices, numNodesAtDepth*4*sizeof(int), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(edgePlacement_device, ownerInidices, numNodesAtDepth*4*sizeof(int), hipMemcpyHostToDevice));
    delete[] ownerInidices;

    prevCount = numEdges;
    edgeDepthIndex_host[i] = numEdges;

    findEdgeOwners<T><<<grid, block>>>(numNodesAtDepth, this->nodeDepthIndex->host[i], this->nodes->device, atomicCounter, ownerInidices_device, edgePlacement_device);
    CudaCheckError();
    CudaSafeCall(hipMemcpy(&numEdges, atomicCounter, sizeof(int), hipMemcpyDeviceToHost));
    if(i == this->depth  && numEdges - prevCount != 4){
      std::cout<<"ERROR GENERATING EDGES, vertices at depth 0 != 4 -> "<<numEdges - prevCount<<std::endl;
      exit(-1);
    }

    CudaSafeCall(hipMalloc((void**)&edges2D[i], (numEdges - prevCount)*sizeof(Edge)));
    CudaSafeCall(hipMalloc((void**)&compactedOwnerArray_device,(numEdges - prevCount)*sizeof(int)));
    CudaSafeCall(hipMalloc((void**)&compactedEdgePlacement_device,(numEdges - prevCount)*sizeof(int)));

    thrust::device_ptr<int> arrayToCompact(ownerInidices_device);
    thrust::device_ptr<int> arrayOut(compactedOwnerArray_device);
    thrust::device_ptr<int> placementToCompact(edgePlacement_device);
    thrust::device_ptr<int> placementOut(compactedEdgePlacement_device);

    thrust::copy_if(arrayToCompact, arrayToCompact + (numNodesAtDepth*4), arrayOut, is_not_neg());
    CudaCheckError();
    thrust::copy_if(placementToCompact, placementToCompact + (numNodesAtDepth*4), placementOut, is_not_neg());
    CudaCheckError();

    CudaSafeCall(hipFree(ownerInidices_device));
    CudaSafeCall(hipFree(edgePlacement_device));

    //reset and allocated resources
    grid = {1,1,1};
    block = {1,1,1};
    getGrid(numEdges - prevCount, grid);


    fillUniqueEdgeArray<T><<<grid, block>>>(this->nodeDepthIndex->host[i], this->nodes->device, numEdges - prevCount,
      edgeDepthIndex_host[i], edges2D[i], this->depth - i, compactedOwnerArray_device, compactedEdgePlacement_device);
    CudaCheckError();
    CudaSafeCall(hipFree(compactedOwnerArray_device));
    CudaSafeCall(hipFree(compactedEdgePlacement_device));

  }
  Edge* edges_device;
  CudaSafeCall(hipMalloc((void**)&edges_device, numEdges*sizeof(Edge)));
  for(int i = 0; i <= this->depth; ++i){
    if(i < this->depth){
      CudaSafeCall(hipMemcpy(edges_device + edgeDepthIndex_host[i], edges2D[i], (edgeDepthIndex_host[i+1] - edgeDepthIndex_host[i])*sizeof(Edge), hipMemcpyDeviceToDevice));
    }
    else{
      CudaSafeCall(hipMemcpy(edges_device + edgeDepthIndex_host[i], edges2D[i], 4*sizeof(Edge), hipMemcpyDeviceToDevice));
    }
    CudaSafeCall(hipFree(edges2D[i]));
  }
  CudaSafeCall(hipFree(edges2D_device));

  this->edges = new Unity<Edge>(edges_device, numEdges, gpu);
  this->edgeDepthIndex = new Unity<unsigned int>(edgeDepthIndex_host, this->depth + 1, cpu);

}

template<typename T>
void jax::Quadtree<T>::generateVerticesAndEdges(){
  this->generateVertices();
  this->generateEdges();
}

template<typename T>
void jax::Quadtree<T>::setNodeFlags(jax::Unity<bool>* hashMap, bool requireFullNeighbors, uint2 depthRange){
  if(hashMap == nullptr || hashMap->state == null){
    throw NullUnityException("hashMap must be filled before setFlags is called");
  }
  if(!(depthRange.x == 0 && depthRange.y == 0) && (depthRange.x > depthRange.y || depthRange.x > this->depth || this->depth > depthRange.y || this->depth < depthRange.x)){
    std::cout<<"ERROR: invalid depthRange in setFlags"<<std::endl;
    exit(-1);
  }
  MemoryState origin[3] = {hashMap->state,this->nodes->state,this->nodeDepthIndex->state};
  if(hashMap->fore == cpu) hashMap->transferMemoryTo(gpu);
  if(this->nodes->fore == cpu) this->nodes->transferMemoryTo(gpu);
  if(this->nodeDepthIndex->fore == gpu) this->nodeDepthIndex->transferMemoryTo(cpu);

  unsigned int nodeDepthIndex = 0;
  if(depthRange.y == 0){
    nodeDepthIndex = this->nodeDepthIndex->host[0];
  }
  else{
    nodeDepthIndex = this->nodeDepthIndex->host[this->depth - depthRange.y];
  }
  unsigned int numNodes = 0;
  if(depthRange.x == 0){
    numNodes = this->nodes->numElements - nodeDepthIndex;
  }
  else{
    numNodes = this->nodeDepthIndex->host[this->depth - (depthRange.x - 1)] - nodeDepthIndex;
  }

  dim3 grid = {1,1,1};
  dim3 block = {1,1,1};
  getFlatGridBlock(numNodes, grid, block);

  applyNodeFlags<T><<<grid,block>>>(numNodes,nodeDepthIndex,this->nodes->device,hashMap->device,requireFullNeighbors);
  hipDeviceSynchronize();
  CudaCheckError();

  if(origin[0] != hashMap->state){
    hashMap->setMemoryState(origin[0]);
  }
  this->nodes->fore = gpu;//due to editing nodes in this method
  if(origin[1] != this->nodes->state){
    this->nodes->setMemoryState(origin[1]);
  }
  if(origin[2] != this->nodeDepthIndex->state){
    this->nodeDepthIndex->setMemoryState(origin[2]);
  }
}
template<typename T>
void jax::Quadtree<T>::setNodeFlags(float2 flagBorder, bool requireFullNeighbors, uint2 depthRange){
  if(!(depthRange.x == 0 && depthRange.y == 0) && (depthRange.x > depthRange.y || depthRange.x > this->depth || this->depth > depthRange.y ||
    this->depth < depthRange.x)){
    std::cout<<"ERROR: invalid depthRange in setFlags"<<std::endl;
    exit(-1);
  }
  MemoryState origin[2] = {this->nodes->state,this->nodeDepthIndex->state};
  if(this->nodes->fore == cpu) this->nodes->transferMemoryTo(gpu);
  if(this->nodeDepthIndex->fore == gpu) this->nodeDepthIndex->transferMemoryTo(cpu);

  unsigned int nodeDepthIndex = 0;
  if(depthRange.y == 0){
    nodeDepthIndex = this->nodeDepthIndex->host[0];
  }
  else{
    nodeDepthIndex = this->nodeDepthIndex->host[this->depth - depthRange.y];
  }
  unsigned int numNodes = 0;
  if(depthRange.x == 0){
    numNodes = this->nodes->numElements - nodeDepthIndex;
  }
  else{
    numNodes = this->nodeDepthIndex->host[this->depth - (depthRange.x - 1)] - nodeDepthIndex;
  }

  dim3 grid = {1,1,1};
  dim3 block = {1,1,1};
  getFlatGridBlock(numNodes, grid, block);

  printf("Setting node flags based on distance from edge = {%f,%f} ",flagBorder.x,flagBorder.y);
  if(requireFullNeighbors)printf("while also requiring full neighbors");
  printf("\n");
  float4 bounds = {flagBorder.x, flagBorder.y, ((float)this->size.x) - flagBorder.x, ((float)this->size.y) - flagBorder.y};

  applyNodeFlags<T><<<grid,block>>>(numNodes,nodeDepthIndex,this->nodes->device,bounds,requireFullNeighbors);
  hipDeviceSynchronize();
  CudaCheckError();

  this->nodes->fore = gpu;//due to editing nodes in this method
  if(origin[0] != this->nodes->state){
    this->nodes->setMemoryState(origin[0]);
  }
  if(origin[1] != this->nodeDepthIndex->state){
    this->nodeDepthIndex->setMemoryState(origin[1]);
  }
}

template<typename T>
void jax::Quadtree<T>::writePLY(){
  std::string newFile = "out/test_"+ std::to_string(rand())+ ".ply";
  std::cout<<"writing "<<newFile<<std::endl;
  std::ofstream plystream(newFile);
  if (plystream.is_open()) {
    int verticesToWrite = this->nodes->numElements;
    this->nodes->transferMemoryTo(cpu);
    std::ostringstream stringBuffer = std::ostringstream("");
    stringBuffer << "ply\nformat ascii 1.0\ncomment object: SSRL test\n";
    stringBuffer << "element vertex ";
    stringBuffer << verticesToWrite;
    stringBuffer << "\nproperty float x\nproperty float y\nproperty float z\n";
    stringBuffer << "property uchar red\nproperty uchar green\nproperty uchar blue\n";
    stringBuffer << "end_header\n";
    plystream << stringBuffer.str();
    for(int i = 0; i < verticesToWrite; ++i){
      stringBuffer = std::ostringstream("");
      stringBuffer << this->nodes->host[i].center.x;
      stringBuffer << " ";
      stringBuffer << this->nodes->host[i].center.y;
      stringBuffer << " 0 ";
      if(this->nodes->host[i].flag){
        stringBuffer << " 0 0 0\n";
      }
      else{
        stringBuffer << " 255 255 255\n";
      }
      plystream << stringBuffer.str();
    }

    std::cout<<newFile + " has been created.\n"<<std::endl;
  }
  else{
    std::cout << "Unable to open: " + newFile<< std::endl;
    exit(1);
  }
}
template<>
void jax::Quadtree<unsigned char>::writePLY(){
std::string newFile = "out/test_"+ std::to_string(rand())+ ".ply";
std::ofstream plystream(newFile);
if (plystream.is_open()) {
  int verticesToWrite = this->nodes->numElements;
  this->nodes->transferMemoryTo(cpu);
  this->data->transferMemoryTo(cpu);
  std::ostringstream stringBuffer = std::ostringstream("");
  stringBuffer << "ply\nformat ascii 1.0\ncomment object: SSRL test\n";
  stringBuffer << "element vertex ";
  stringBuffer << verticesToWrite;
  stringBuffer << "\nproperty float x\nproperty float y\nproperty float z\n";
  stringBuffer << "property uchar red\nproperty uchar green\nproperty uchar blue\n";
  stringBuffer << "end_header\n";
  plystream << stringBuffer.str();
  for(int i = 0; i < verticesToWrite; ++i){
    stringBuffer = std::ostringstream("");
    stringBuffer << this->nodes->host[i].center.x;
    stringBuffer << " ";
    stringBuffer << this->nodes->host[i].center.y;
    stringBuffer << " ";
    stringBuffer << this->nodes->host[i].depth*1000;
    stringBuffer << " ";
    int color = 0;
    if(this->nodes->host[i].numElements != 0 && this->nodes->host[i].flag){
      int numNodes = 0;
      int index = this->nodes->host[i].dataIndex;
      if(index == -1){
        std::cout<<"ERROR: node with "<<this->nodes->host[i].numElements<<" elements has dataIndex of -1 at depth "<<this->nodes->host[i].depth
        <<" "<<this->nodes->host[i].center.x<<","<<this->nodes->host[i].center.y<<std::endl;
        exit(-1);
      }
      for(int c = index; c < index + this->nodes->host[i].numElements; ++c){
        color += (int) this->data->host[c];
        numNodes++;
      }
      if(numNodes > 1) color /= numNodes;
    }
    stringBuffer << color;
    stringBuffer << " ";
    stringBuffer << color;
    stringBuffer << " ";
    stringBuffer << color;
    stringBuffer << "\n";
    plystream << stringBuffer.str();
  }

  std::cout<<newFile + " has been created.\n"<<std::endl;
}
else{
  std::cout << "Unable to open: " + newFile<< std::endl;
  exit(1);
}
}
template<>
void jax::Quadtree<unsigned int>::writePLY(jax::Unity<unsigned char>* pixels){
  std::string newFile = "out/test_"+ std::to_string(rand())+ ".ply";
  std::ofstream plystream(newFile);
  if (plystream.is_open()) {
    int verticesToWrite = this->nodes->numElements;
    this->nodes->transferMemoryTo(cpu);
    pixels->transferMemoryTo(cpu);
    this->data->transferMemoryTo(cpu);
    std::ostringstream stringBuffer = std::ostringstream("");
    stringBuffer << "ply\nformat ascii 1.0\ncomment object: SSRL test\n";
    stringBuffer << "element vertex ";
    stringBuffer << verticesToWrite;
    stringBuffer << "\nproperty float x\nproperty float y\nproperty float z\n";
    stringBuffer << "property uchar red\nproperty uchar green\nproperty uchar blue\n";
    stringBuffer << "end_header\n";
    plystream << stringBuffer.str();
    for(int i = 0; i < verticesToWrite; ++i){
      stringBuffer = std::ostringstream("");
      stringBuffer << this->nodes->host[i].center.x;
      stringBuffer << " ";
      stringBuffer << this->nodes->host[i].center.y;
      stringBuffer << " 0 ";
      int color = 0;
      if(this->nodes->host[i].numElements != 0){
        int numNodes = 0;
        int index = this->data->host[this->nodes->host[i].dataIndex];
        if(index == -1){
          std::cout<<"ERROR node"<<std::endl;
        }
        for(int c = index; c < index + this->nodes->host[i].numElements; ++c){
          color += (int) pixels->host[c];
          numNodes++;
        }
        if(numNodes > 1) color /= numNodes;
      }
      stringBuffer << color;
      stringBuffer << " 0 0";
      stringBuffer << "\n";
      plystream << stringBuffer.str();
    }

    std::cout<<newFile + " has been created.\n"<<std::endl;
  }
  else{
    std::cout << "Unable to open: " + newFile<< std::endl;
    exit(1);
  }
}

template<typename T>
void jax::Quadtree<T>::writePLY(Node* nodes_device, unsigned long numNodes){
  std::string newFile = "out/test_"+ std::to_string(rand())+ ".ply";
  std::ofstream plystream(newFile);
  if (plystream.is_open()) {
    int verticesToWrite = numNodes;

    Node* nodes_host = new Node[numNodes];
    CudaSafeCall(hipMemcpy(nodes_host,nodes_device,numNodes*sizeof(Node),hipMemcpyDeviceToHost));
    std::ostringstream stringBuffer = std::ostringstream("");
    stringBuffer << "ply\nformat ascii 1.0\ncomment object: SSRL test\n";
    stringBuffer << "element vertex ";
    stringBuffer << verticesToWrite;
    stringBuffer << "\nproperty float x\nproperty float y\nproperty float z\n";
    stringBuffer << "end_header\n";
    plystream << stringBuffer.str();
    for(int i = 0; i < verticesToWrite; ++i){
      stringBuffer = std::ostringstream("");
      stringBuffer << nodes_host[i].center.x;
      stringBuffer << " ";
      stringBuffer << nodes_host[i].center.y;
      stringBuffer << " 0.0";
      stringBuffer << "\n";
      plystream << stringBuffer.str();
    }
    delete[] nodes_host;
    std::cout<<newFile + " has been created.\n"<<std::endl;
  }
  else{
    std::cout << "Unable to open: " + newFile<< std::endl;
    exit(1);
  }
}

template<typename T>
void jax::Quadtree<T>::writePLY(float2* points_device, unsigned long numPoints){
  std::string newFile = "out/test_"+ std::to_string(rand())+ ".ply";
  std::ofstream plystream(newFile);
  if (plystream.is_open()) {
    int verticesToWrite = numPoints;

    float2* points_host = new float2[numPoints];
    CudaSafeCall(hipMemcpy(points_host,points_device,numPoints*sizeof(float2),hipMemcpyDeviceToHost));
    std::ostringstream stringBuffer = std::ostringstream("");
    stringBuffer << "ply\nformat ascii 1.0\ncomment object: SSRL test\n";
    stringBuffer << "element vertex ";
    stringBuffer << verticesToWrite;
    stringBuffer << "\nproperty float x\nproperty float y\nproperty float z\n";
    stringBuffer << "end_header\n";
    plystream << stringBuffer.str();
    for(int i = 0; i < verticesToWrite; ++i){
      stringBuffer = std::ostringstream("");
      stringBuffer << points_host[i].x;
      stringBuffer << " ";
      stringBuffer << points_host[i].y;
      stringBuffer << " 0.0";
      stringBuffer << "\n";
      plystream << stringBuffer.str();
    }
    delete[] points_host;
    std::cout<<newFile + " has been created.\n"<<std::endl;
  }
  else{
    std::cout << "Unable to open: " + newFile<< std::endl;
    exit(1);
  }
}



/*
CUDA implementations
*/
//NOTE: THIS SHOULD ONLY BE USED FOR DENSE POINTER QUADTREE
__global__ void jax::getKeys(int* keys, float2* nodeCenters, uint2 size, int2 border, unsigned int depth){
  unsigned int globalID = (blockIdx.y* gridDim.x+ blockIdx.x)* blockDim.x + threadIdx.x;
  if(globalID < (size.x - (border.x*2))*(size.y - (border.y*2))){
    float x = ((float)((globalID%(size.x - (border.x*2))) + border.x)) + 0.5f;
    float y = ((float)((globalID/(size.x - (border.x*2))) + border.y)) + 0.5f;
    int key = 0;
    unsigned int depth_reg = depth;
    int currentDepth = 1;
    float2 reg_size = {((float)size.x)/2.0f, ((float)size.y)/2.0f};
    float2 center = reg_size;
    while(depth_reg >= currentDepth){
      reg_size.x /= 2.0f;
      reg_size.y /= 2.0f;
      currentDepth++;
      if(x < center.x){
        key <<= 1;
        center.x -= reg_size.x;
      }
      else{
        key = (key << 1) + 1;
        center.x += reg_size.x;
      }
      if(y < center.y){
        key <<= 1;
        center.y -= reg_size.y;
      }
      else{
        key = (key << 1) + 1;
        center.y += reg_size.y;
      }
    }
    keys[globalID] = key;
    nodeCenters[globalID] = center;
  }
}
__global__ void jax::getKeys(int* keys, float2* nodeCenters, uint2 size, int2 border, unsigned int depth, unsigned int colorDepth){
  unsigned int blockID = blockIdx.y* gridDim.x+ blockIdx.x;
  if(blockID < (size.x - (border.x*2))*(size.y - (border.y*2))*colorDepth){
    float x = ((float)((blockID%(size.x - (border.x*2))) + border.x)) + 0.5f;
    float y = ((float)((blockID/(size.x - (border.x*2))) + border.y)) + 0.5f;
    int key = 0;
    unsigned int depth_reg = depth;
    int currentDepth = 1;
    float2 reg_size = {((float)size.x)/2.0f, ((float)size.y)/2.0f};
    float2 center = reg_size;
    while(depth_reg >= currentDepth){
      reg_size.x /= 2.0f;
      reg_size.y /= 2.0f;
      currentDepth++;
      if(x < center.x){
        key <<= 1;
        center.x -= reg_size.x;
      }
      else{
        key = (key << 1) + 1;
        center.x += reg_size.x;
      }
      if(y < center.y){
        key <<= 1;
        center.y -= reg_size.y;
      }
      else{
        key = (key << 1) + 1;
        center.y += reg_size.y;
      }
    }
    keys[blockID*colorDepth + threadIdx.x] = key;
    nodeCenters[blockID*colorDepth + threadIdx.x] = center;
  }
}
__global__ void jax::getKeys(unsigned int numPoints, float2* points, int* keys, float2* nodeCenters, uint2 size, unsigned int depth){
  unsigned int globalID = (blockIdx.y* gridDim.x+ blockIdx.x)* blockDim.x + threadIdx.x;
  if(globalID < numPoints){
    float2 point = points[globalID];
    int key = 0;
    unsigned int depth_reg = depth;
    int currentDepth = 1;
    float2 reg_size = {((float)size.x)/2.0f, ((float)size.y)/2.0f};
    float2 center = reg_size;
    while(depth_reg >= currentDepth){
      reg_size.x /= 2.0f;
      reg_size.y /= 2.0f;
      currentDepth++;
      if(point.x < center.x){
        key <<= 1;
        center.x -= reg_size.x;
      }
      else{
        key = (key << 1) + 1;
        center.x += reg_size.x;
      }
      if(point.y < center.y){
        key <<= 1;
        center.y -= reg_size.y;
      }
      else{
        key = (key << 1) + 1;
        center.y += reg_size.y;
      }
    }
    keys[globalID] = key;
    nodeCenters[globalID] = center;
  }
}
__global__ void jax::getKeys(unsigned int numLocalizedPointers, jax::LocalizedData<unsigned int>* localizedPointers, int* keys, float2* nodeCenters, uint2 size, unsigned int depth){
  unsigned int globalID = (blockIdx.y* gridDim.x+ blockIdx.x)* blockDim.x + threadIdx.x;
  if(globalID < numLocalizedPointers){
    float2 point = localizedPointers[globalID].loc;
    int key = 0;
    unsigned int depth_reg = depth;
    int currentDepth = 1;
    float2 reg_size = {((float)size.x)/2.0f, ((float)size.y)/2.0f};
    float2 center = reg_size;
    while(depth_reg >= currentDepth){
      reg_size.x /= 2.0f;
      reg_size.y /= 2.0f;
      currentDepth++;
      if(point.x < center.x){
        key <<= 1;
        center.x -= reg_size.x;
      }
      else{
        key = (key << 1) + 1;
        center.x += reg_size.x;
      }
      if(point.y < center.y){
        key <<= 1;
        center.y -= reg_size.y;
      }
      else{
        key = (key << 1) + 1;
        center.y += reg_size.y;
      }
    }
    keys[globalID] = key;
    nodeCenters[globalID] = center;
  }
}


template<typename T>
__global__ void jax::fillLeafNodes(unsigned long numDataElements, unsigned long numLeafNodes, typename jax::Quadtree<T>::Node* leafNodes,
int* keys, float2* nodeCenters, unsigned int* nodeDataIndex, unsigned int depth){
  unsigned int globalID = (blockIdx.y* gridDim.x+ blockIdx.x)* blockDim.x + threadIdx.x;
  if(globalID < numLeafNodes){
    typename Quadtree<T>::Node node = typename Quadtree<T>::Node();
    node.key = keys[globalID];
    node.dataIndex = nodeDataIndex[globalID];
    if(globalID + 1 != numLeafNodes){
      node.numElements = nodeDataIndex[globalID + 1] - node.dataIndex;
    }
    else{
      node.numElements = numDataElements - node.dataIndex;
    }
    node.center = nodeCenters[node.dataIndex];//centers are not compacted by key so
    node.depth = depth;
    leafNodes[globalID] = node;
  }
}


template<typename T>
__global__ void jax::findAllNodes(unsigned long numUniqueNodes, unsigned int* nodeNumbers, typename jax::Quadtree<T>::Node* uniqueNodes){
  unsigned int globalID = (blockIdx.y* gridDim.x+ blockIdx.x)* blockDim.x + threadIdx.x;
  int tempCurrentKey = 0;
  int tempPrevKey = 0;
  if(globalID < numUniqueNodes){
    if(globalID == 0){
      nodeNumbers[globalID] = 0;
      return;
    }
    tempCurrentKey = uniqueNodes[globalID].key>>2;
    tempPrevKey = uniqueNodes[globalID - 1].key>>2;
    if(tempPrevKey == tempCurrentKey){
      nodeNumbers[globalID] = 0;
    }
    else{
      nodeNumbers[globalID] = 4;
    }
  }
}

template<typename T>
__global__ void jax::fillNodesAtDepth(unsigned long numUniqueNodes, unsigned int* nodeNumbers, unsigned int* nodeAddresses, typename jax::Quadtree<T>::Node* existingNodes,
typename jax::Quadtree<T>::Node* allNodes, unsigned int currentDepth, unsigned int totalDepth){
  unsigned int globalID = (blockIdx.y* gridDim.x+ blockIdx.x)* blockDim.x + threadIdx.x;
  if(currentDepth != 0 && globalID < numUniqueNodes){
    typename Quadtree<T>::Node node = existingNodes[globalID];
    unsigned int address = nodeAddresses[globalID];
    allNodes[address + (node.key&0x00000003)] = node;
    if(nodeNumbers[globalID] == 4 || globalID == 0){
      int siblingKey = node.key&0xfffffffc;//will clear last 2 bits
      for(int i = 0; i < 4; ++i){
        allNodes[address + i].depth = currentDepth;
        allNodes[address + i].key = siblingKey + i;
      }
    }
  }
  else if(currentDepth == 0){
    allNodes[nodeAddresses[0]] = existingNodes[0];
  }
}

template<typename T>
__global__ void jax::buildParentalNodes(unsigned long numChildNodes, unsigned long childDepthIndex, typename jax::Quadtree<T>::Node* childNodes, typename jax::Quadtree<T>::Node* parentNodes, uint2 size){
  unsigned long numUniqueNodesAtParentDepth = numChildNodes / 4;
  unsigned int globalID = (blockIdx.y* gridDim.x+ blockIdx.x)* blockDim.x + threadIdx.x;
  int nodesIndex = globalID*4;
  int2 childLoc[4] = {
    {-1,-1},
    {-1,1},
    {1,-1},
    {1,1}
  };
  if(globalID < numUniqueNodesAtParentDepth){
    typename Quadtree<T>::Node node = typename Quadtree<T>::Node();//may not be necessary
    node.key = (childNodes[nodesIndex].key>>2);
    node.depth =  childNodes[nodesIndex].depth - 1;

    float2 widthOfNode = {((float)size.x)/powf(2,node.depth + 1),((float)size.y)/powf(2,node.depth + 1)};

    int chosen = 0;
    for(int i = 0; i < 4; ++i){
      if(childNodes[nodesIndex + i].dataIndex != -1){
        if(node.dataIndex == -1){
          node.dataIndex = childNodes[nodesIndex + i].dataIndex;
          node.center.x = childNodes[nodesIndex + i].center.x - (widthOfNode.x*0.5*childLoc[i].x);
          node.center.y = childNodes[nodesIndex + i].center.y - (widthOfNode.y*0.5*childLoc[i].y);
          chosen = i;
        }
        node.numElements += childNodes[nodesIndex + i].numElements;
      }
      node.children[i] = nodesIndex + childDepthIndex + i;
    }
    for(int i = 0; i < 4; ++i){
      if(childNodes[nodesIndex + i].dataIndex == -1){
        childNodes[nodesIndex + i].center.x = node.center.x + (widthOfNode.x*0.5*childLoc[i].x);
        childNodes[nodesIndex + i].center.y = node.center.y + (widthOfNode.y*0.5*childLoc[i].y);
      }
      if(childNodes[nodesIndex + i].center.x - (widthOfNode.x*0.5*childLoc[i].x) != node.center.x || childNodes[nodesIndex + i].center.y - (widthOfNode.y*0.5*childLoc[i].y) != node.center.y){
        printf("%f,%f = %f,%f (%f,%f)%d,%d\n",node.center.x,node.center.y,childNodes[nodesIndex+i].center.x,childNodes[nodesIndex+i].center.y,childNodes[nodesIndex+chosen].center.x,childNodes[nodesIndex+chosen].center.y,node.depth + 1,node.dataIndex);
        //asm("trap;");
      }
    }
    parentNodes[globalID] = node;
  }
}

//NOTE this is recursive
template<typename T>
__global__ void jax::fillParentIndex(unsigned int numNodesAtDepth, unsigned int depthStartingIndex, typename Quadtree<T>::Node* nodes){
  unsigned int blockID = blockIdx.y * gridDim.x + blockIdx.x;
  if(blockID < numNodesAtDepth && nodes[blockID + depthStartingIndex].children[threadIdx.x] != -1){
    nodes[nodes[blockID + depthStartingIndex].children[threadIdx.x]].parent = depthStartingIndex + blockID;
  }
}

template<typename T>
__global__ void jax::fillDataNodeIndex(unsigned long numLeafNodes, typename Quadtree<T>::Node* nodes, unsigned int* dataNodeIndex){
  unsigned int globalID = (blockIdx.y* gridDim.x+ blockIdx.x) * blockDim.x + threadIdx.x;
  if(globalID < numLeafNodes){//no need for depth index as leaf nodes come first in node ordering
    typename Quadtree<T>::Node node = nodes[globalID];
    for(int i = 0;node.dataIndex != -1 && i < node.numElements; ++i){
      dataNodeIndex[node.dataIndex + i] = globalID;
    }
  }
}

template<typename T>
__global__ void jax::computeNeighboringNodes(unsigned int numNodesAtDepth, unsigned int currentDepthIndex, unsigned int* parentLUT,
unsigned int* childLUT, typename Quadtree<T>::Node* nodes){
  unsigned int blockID = blockIdx.y * gridDim.x + blockIdx.x;
  if(blockID < numNodesAtDepth){
    int neighborParentIndex = 0;
    nodes[blockID + currentDepthIndex].neighbors[4] = blockID + currentDepthIndex;
    __syncthreads();//threads wait until all other threads have finished above operations
    if(nodes[blockID + currentDepthIndex].parent != -1){
      int parentIndex = nodes[blockID + currentDepthIndex].parent;
      int depthKey = nodes[blockID + currentDepthIndex].key&(0x00000003);//will clear all but last 2 bits
      int lutIndexHelper = (depthKey*9) + threadIdx.x;
      int parentLUTIndex = parentLUT[lutIndexHelper];
      int childLUTIndex = childLUT[lutIndexHelper];
      neighborParentIndex = nodes[parentIndex].neighbors[parentLUTIndex];
      if(neighborParentIndex != -1){
        nodes[blockID + currentDepthIndex].neighbors[threadIdx.x] = nodes[neighborParentIndex].children[childLUTIndex];
      }
    }
  }
}

template<typename T>
__global__ void jax::findVertexOwners(unsigned int numNodesAtDepth, unsigned int depthIndex, typename jax::Quadtree<T>::Node* nodes, int* numVertices, int* ownerInidices, int* vertexPlacement){
  unsigned int vertexLUT[4][3] = {
    {0,1,3},
    {1,2,5},
    {3,6,7},
    {5,7,8}
  };
  unsigned int blockID = blockIdx.y * gridDim.x + blockIdx.x;
  if(blockID < numNodesAtDepth){
    int vertexID = (blockID*4) + threadIdx.x;
    int sharesVertex = -1;
    for(int i = 0; i < 3; ++i){//iterate through neighbors that share edge
      sharesVertex = vertexLUT[threadIdx.x][i];
      if(nodes[blockID + depthIndex].neighbors[sharesVertex] != -1 && sharesVertex < 4){//less than itself
        return;
      }
    }
    //if thread reaches this point, that means that this edge is owned by the current node
    //also means owner == current node
    ownerInidices[vertexID] = blockID + depthIndex;
    vertexPlacement[vertexID] = threadIdx.x;
    atomicAdd(numVertices, 1);
  }
}

template<typename T>
__global__ void jax::fillUniqueVertexArray(unsigned int depthIndex, typename jax::Quadtree<T>::Node* nodes, unsigned long numVertices, int vertexIndex,
typename jax::Quadtree<T>::Vertex* vertices, int depth, int* ownerInidices, int* vertexPlacement, uint2 size){
  unsigned int vertexLUT[4][3] = {
    {0,1,3},
    {1,2,5},
    {3,6,7},
    {5,7,8}
  };
  int2 coordPlacementIdentity[4] = {
    {-1,-1},
    {-1,1},
    {1,-1},
    {1,1}
  };
  unsigned int globalID = (blockIdx.y* gridDim.x+ blockIdx.x)* blockDim.x + threadIdx.x;
  if(globalID < numVertices){

    int ownerNodeIndex = ownerInidices[globalID];
    int ownedIndex = vertexPlacement[globalID];

    nodes[ownerNodeIndex].vertices[ownedIndex] = globalID + vertexIndex;

    float2 depthHalfWidth = {size.x/powf(2, depth + 1),size.y/powf(2, depth + 1)};
    typename Quadtree<T>::Vertex vertex = typename Quadtree<T>::Vertex();
    vertex.loc.x = nodes[ownerNodeIndex].center.x + (depthHalfWidth.x*coordPlacementIdentity[ownedIndex].x);
    vertex.loc.y = nodes[ownerNodeIndex].center.y + (depthHalfWidth.y*coordPlacementIdentity[ownedIndex].y);

    vertex.depth = depth;
    vertex.nodes[0] = ownerNodeIndex;
    int neighborSharingVertex = -1;
    for(int i = 0; i < 3; ++i){
      neighborSharingVertex = nodes[ownerNodeIndex].neighbors[vertexLUT[ownedIndex][i]];
      vertex.nodes[i + 1] =  neighborSharingVertex;
      if(neighborSharingVertex == -1) continue;
      //WARNING CHECK THIS
      nodes[neighborSharingVertex].vertices[2 - i] = globalID + vertexIndex;
    }
    vertices[globalID] = vertex;
  }
}

template<typename T>
__global__ void jax::findEdgeOwners(unsigned int numNodesAtDepth, unsigned int depthIndex, typename jax::Quadtree<T>::Node* nodes, int* numEdges, int* ownerInidices, int* edgePlacement){
  unsigned int edgeLUT[4] = {1,3,5,7};
  unsigned blockID = blockIdx.y * gridDim.x + blockIdx.x;
  if(blockID < numNodesAtDepth){
    int edgeID = (blockID*4) + threadIdx.x;
    int sharesEdge = -1;
    sharesEdge = edgeLUT[threadIdx.x];
    if(nodes[blockID + depthIndex].neighbors[sharesEdge] != -1 && sharesEdge < 4){//less than itself
      return;
    }
    //if thread reaches this point, that means that this edge is owned by the current node
    //also means owner == current node
    ownerInidices[edgeID] = blockID + depthIndex;
    edgePlacement[edgeID] = threadIdx.x;
    atomicAdd(numEdges, 1);
  }
}

template<typename T>
__global__ void jax::fillUniqueEdgeArray(unsigned int depthIndex, typename jax::Quadtree<T>::Node* nodes, unsigned long numEdges, int edgeIndex,
typename jax::Quadtree<T>::Edge* edges, int depth, int* ownerInidices, int* edgePlacement){
  unsigned int edgeLUT[4] = {1,3,5,7};
  uint2 vertexEdgeIdentity[4] = {
    {0,1},
    {0,2},
    {1,3},
    {3,2}
  };
  unsigned long blockId = blockIdx.y* gridDim.x+ blockIdx.x;
  unsigned long globalID = blockId * blockDim.x + threadIdx.x;
  if(globalID < numEdges){
    int ownerNodeIndex = ownerInidices[globalID];
    int ownedIndex = edgePlacement[globalID];

    nodes[ownerNodeIndex].edges[ownedIndex] = globalID + edgeIndex;

    typename Quadtree<T>::Edge edge = typename Quadtree<T>::Edge();
    edge.vertices.x = nodes[ownerNodeIndex].vertices[vertexEdgeIdentity[ownedIndex].x];
    edge.vertices.y = nodes[ownerNodeIndex].vertices[vertexEdgeIdentity[ownedIndex].y];
    edge.depth = depth;
    edge.nodes[0] = ownerNodeIndex;

    int neighborSharingFace = -1;
    neighborSharingFace = nodes[ownerNodeIndex].neighbors[edgeLUT[ownedIndex]];
    edge.nodes[1] =  neighborSharingFace;
    if(neighborSharingFace != -1) nodes[neighborSharingFace].edges[3 - ownedIndex] = globalID + edgeIndex;
    edges[globalID] = edge;
  }
}

template<typename T>
__global__ void jax::applyNodeFlags(unsigned int numNodes, unsigned int depthIndex, typename jax::Quadtree<T>::Node* nodes, bool* hashMap, bool requireFullNeighbors){
  unsigned int globalID = (blockIdx.y* gridDim.x+ blockIdx.x)* blockDim.x + threadIdx.x;
  if(globalID < numNodes){
    typename Quadtree<T>::Node node = nodes[globalID + depthIndex];
    node.flag = hashMap[globalID];
    if(requireFullNeighbors && !node.flag){
      for(int i = 0; i < 9; ++i){
        if(node.neighbors[i] == -1){
          nodes[globalID + depthIndex].flag = false;
          return;
        }
      }
    }
  }
}
template<typename T>
__global__ void jax::applyNodeFlags(unsigned int numNodes, unsigned int depthIndex, typename jax::Quadtree<T>::Node* nodes, float4 flagBounds, bool requireFullNeighbors){
  unsigned int globalID = (blockIdx.y* gridDim.x+ blockIdx.x)* blockDim.x + threadIdx.x;
  if(globalID < numNodes){
    globalID += depthIndex;
    typename Quadtree<T>::Node node = nodes[globalID];
    if(node.center.x > flagBounds.x && node.center.y > flagBounds.y && node.center.x < flagBounds.z && node.center.y < flagBounds.w){
      if(requireFullNeighbors){
        for(int i = 0; i < 9; ++i){
          if(node.neighbors[i] == -1){
            nodes[globalID].flag = false;
            return;
          }
        }
      }
      nodes[globalID].flag = true;
    }
    else{
      nodes[globalID].flag = false;
    }
  }
}
